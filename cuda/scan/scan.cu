#include "hip/hip_runtime.h"
#include <iostream>

__global__ void scan(float *g_odata, float *g_idata, int n)  {  
    extern __shared__ float temp[]; 

    int thid = threadIdx.x;
    int offset = 1;
    temp[2*thid] = g_idata[2*thid]; 
    temp[2*thid+1] = g_idata[2*thid+1];  
	
    for (int d = n >> 1; d > 0; d >>= 1) {
	
	__syncthreads();  
	if (thid < d) {

	    int ai = offset*(2*thid+1)-1;  
	    int bi = offset*(2*thid+2)-1;  	    
	    temp[bi] += temp[ai];  
	}	
	offset *= 2;
    }


    if (thid == 0) { temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;}  
 
    for (int d = 1; d < n; d *= 2) {  
	offset >>= 1;  

	__syncthreads();  
	if (thid < d) {
	    int ai = offset*(2*thid+1)-1;  
	    int bi = offset*(2*thid+2)-1;  
	    float t = temp[ai];  
	    temp[ai] = temp[bi];  
	    temp[bi] += t;   
	}
	
    }
    
    __syncthreads();
    
    g_odata[2*thid] = temp[2*thid]; 
    g_odata[2*thid+1] = temp[2*thid+1];  
}


int main() {
    const int n = 10;
    int bytes = n * sizeof(float);
    float *h_a = malloc(bytes), *h_b = malloc(bytes);
    
    for(int i = 0; i < n; i++) {h_a[i] = 1;}
    float * d_a;
    float * d_b;

    hipMalloc((void**)&d_a, bytes);
    hipMalloc((void**)&d_b, bytes);

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    scan<<<1, n, n>>>(d_a, d_b, n);
    hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);


    for(int i = 0; i < n; i++)
	std::cout << h_a[i] << " ";
    std::cout << std::endl;
    for(int i = 0; i < n; i++)
	std::cout << h_b[i] << " ";
    std::cout << std::endl;
    
    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);   
}

